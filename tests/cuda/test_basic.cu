#include "hip/hip_runtime.h"
/**
 * TRACCC library, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

#include <gtest/gtest.h>

TEST(CUDABasic, DeviceCount) {
    int nDev = -1;

    ASSERT_EQ(hipGetDeviceCount(&nDev), hipSuccess);

    ASSERT_GE(nDev, 1);
}

TEST(CUDABasic, Memory) {
    void *ptr = nullptr;

    ASSERT_EQ(hipMalloc(&ptr, 1024), hipSuccess);

    ASSERT_NE(ptr, nullptr);

    ASSERT_EQ(hipFree(ptr), hipSuccess);
}

__global__ void testKernel(int *output) {
    *output = 0x0BADF00D;  // This test sponsored by R1.
}

TEST(CUDABasic, LaunchKernel) {
    int *ptr = nullptr;
    int val = 0;

    ASSERT_EQ(hipMalloc(&ptr, sizeof(int)), hipSuccess);

    ASSERT_NE(ptr, nullptr);

    testKernel<<<1, 1>>>(ptr);

    ASSERT_EQ(hipPeekAtLastError(), hipSuccess);

    ASSERT_EQ(hipMemcpy(&val, ptr, sizeof(int), hipMemcpyDeviceToHost),
              hipSuccess);

    ASSERT_EQ(val, 0x0BADF00D);

    ASSERT_EQ(hipFree(ptr), hipSuccess);
}
